
#include <hip/hip_runtime.h>
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
// All rights reserved. SPDX-License-Identifier: Apache-2.0

// clang-format off

void __device__ __forceinline__ foo() {} // line 6

struct Watermelon {    // line 8
    Watermelon() {}    // line 9
};

template <typename T>                    // line 12
void __device__ __forceinline__ baz() {} // line 13

template <typename T>   // line 15
struct Bax {};          // line 16

enum class Fruit {}; // line 18

typedef Watermelon Suika; // line 20

// clang-format on
