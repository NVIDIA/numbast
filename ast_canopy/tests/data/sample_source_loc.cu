
#include <hip/hip_runtime.h>
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
// All rights reserved. SPDX-License-Identifier: Apache-2.0

// clang-format off

void __device__ __forceinline__ foo() {} // line 6

struct Bar {    // line 8
    Bar() {}    // line 9
};

template <typename T>                    // line 12
void __device__ __forceinline__ baz() {} // line 13

template <typename T>   // line 15
struct Bax {};          // line 16

enum class Watermelon {}; // line 18

typedef Watermelon Suika; // line 20

// clang-format on
